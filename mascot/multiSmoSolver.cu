//
// Created by ss on 16-12-14.
//

#include <sys/time.h>
#include <cfloat>
#include "multiSmoSolver.h"
#include "../svm-shared/constant.h"
#include "hip/hip_runtime.h"
#include "../svm-shared/smoGPUHelper.h"
#include "../svm-shared/HessianIO/deviceHessianOnFly.h"
#include "../SharedUtility/Timer.h"
#include "trainClassifier.h"
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include "../svm-shared/devUtility.h"
#include "../svm-shared/Cache/subHessianCalculator.h"

void MultiSmoSolver::solve() {
    int nrClass = problem.getNumOfClasses();

   if (model.vC.size() == 0) {//initialize C for all the binary classes
        model.vC = vector<float_point>(nrClass * (nrClass - 1) / 2, param.C);
    }

    printf("q = %d, working set size = %d\n", q, workingSetSize);
    //train nrClass*(nrClass-1)/2 binary models
    int k = 0;
	vector<int> prob_start(problem.start);
    for (int i = 0; i < nrClass; ++i) {
        int ci=problem.count[i];
        for (int j = i + 1; j < nrClass; ++j) {
            printf("training classifier with label %d and %d\n", i, j);
            SvmProblem subProblem = problem.getSubProblem(i, j);
            init4Training(subProblem);
            CSRMatrix subProblemMat(subProblem.v_vSamples, subProblem.getNumOfFeatures());
            subProblemMat.copy2Dev(devVal, devRowPtr, devColInd, devSelfDot);
            nnz = subProblemMat.getNnz();
            printf("#positive ins %d, #negative ins %d\n", subProblem.count[0], subProblem.count[1]);
            int totalIter = 0;
            TIMER_START(trainingTimer)
            for (int l = 0;; ++l) {
                if (l == 0) {
                    selectWorkingSetAndPreCompute(subProblem, workingSetSize / 2);
                } else {
                    selectWorkingSetAndPreCompute(subProblem, q / 2);
                }
                TIMER_START(iterationTimer)
                localSMO << < 1, workingSetSize, workingSetSize * sizeof(float) * 3 + 2 * sizeof(float) >> >
                                                 (devLabel, devYiGValue, devAlpha, devAlphaDiff, devWorkingSet, workingSetSize, param.C, devHessianMatrixCache, subProblem.getNumOfSamples());
                TIMER_STOP(iterationTimer)
                TIMER_START(updateGTimer)
                updateF << < gridSize, BLOCK_SIZE >> >
                                       (devYiGValue, devLabel, devWorkingSet, workingSetSize, devAlphaDiff, devHessianMatrixCache, subProblem.getNumOfSamples());
                TIMER_STOP(updateGTimer)
                float diff;
                checkCudaErrors(hipMemcpyFromSymbol(&diff, HIP_SYMBOL(devDiff), sizeof(float_point), 0, hipMemcpyDeviceToHost));
                if (l % 10 == 0)
                    printf(".");
                cout.flush();
                if (diff < EPS) {
                    printf("\nup + low = %f\n", diff);
                    break;
                }
            }
            TIMER_STOP(trainingTimer)
            subProblemMat.freeDev(devVal, devRowPtr, devColInd, devSelfDot);
            vector<int> svIndex;
            vector<float_point> coef;
            float_point rho;
            
            
			extractModel(subProblem, svIndex, coef, rho);
            //countPosiSV(subProblem, svIndex, coef, rho,posi_svcount);
            //model.addBinaryModel(subProblem, svIndex, coef, rho, i, j);
            model.addBinaryLibModel(subProblem, svIndex, coef, rho, i, j, prob_start, ci);
			//******new code for save libmodel
			//int posi_svcount=0;
			//extractLibModel(subProblem, svIndex, coef, rho, nSVi, nSVj, posi_svcount);
	        //model.addBinaryLibModel(subProblem, svIndex, coef, rho, i, j, nSVi, nSVj, model.nonzero, prob_start);
	        //******end new code
	        k++;
            deinit4Training();
        }
    }
}

void MultiSmoSolver::init4Training(const SvmProblem &subProblem) {
    unsigned int trainingSize = subProblem.getNumOfSamples();

    checkCudaErrors(hipMalloc((void **) &devAlpha, sizeof(float_point) * trainingSize));
    checkCudaErrors(hipMalloc((void **) &devYiGValue, sizeof(float_point) * trainingSize));
    checkCudaErrors(hipMalloc((void **) &devLabel, sizeof(int) * trainingSize));

    checkCudaErrors(hipMemset(devAlpha, 0, sizeof(float_point) * trainingSize));
    vector<float_point> negatedLabel(trainingSize);
    for (int i = 0; i < trainingSize; ++i) {
        negatedLabel[i] = -subProblem.v_nLabels[i];
    }
    checkCudaErrors(hipMemcpy(devYiGValue, negatedLabel.data(), sizeof(float_point) * trainingSize,
                               hipMemcpyHostToDevice));
    checkCudaErrors(
            hipMemcpy(devLabel, subProblem.v_nLabels.data(), sizeof(int) * trainingSize, hipMemcpyHostToDevice));

    InitSolver(trainingSize);//initialise base solver

    checkCudaErrors(hipMalloc((void **) &devHessianMatrixCache, sizeof(float_point) * workingSetSize * trainingSize));

    for (int j = 0; j < trainingSize; ++j) {
        hessianDiag[j] = 1;//assume using RBF kernel
    }
    checkCudaErrors(
            hipMemcpy(devHessianDiag, hessianDiag, sizeof(float_point) * trainingSize, hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc((void **) &devFValue4Sort, sizeof(float_point) * trainingSize));
    checkCudaErrors(hipMalloc((void **) &devIdx4Sort, sizeof(int) * trainingSize));

}

void MultiSmoSolver::deinit4Training() {
    checkCudaErrors(hipFree(devAlpha));
    checkCudaErrors(hipFree(devYiGValue));
    checkCudaErrors(hipFree(devLabel));

    DeInitSolver();

    checkCudaErrors(hipFree(devHessianMatrixCache));
    checkCudaErrors(hipFree(devFValue4Sort));
    checkCudaErrors(hipFree(devIdx4Sort));
}

void MultiSmoSolver::extractModel(const SvmProblem &subProblem, vector<int> &svIndex, vector<float_point> &coef,
                                  float_point &rho) const {
    const unsigned int trainingSize = subProblem.getNumOfSamples();
    vector<float_point> alpha(trainingSize);
    const vector<int> &label = subProblem.v_nLabels;
    checkCudaErrors(hipMemcpy(alpha.data(), devAlpha, sizeof(float_point) * trainingSize, hipMemcpyDeviceToHost));
    for (int i = 0; i < trainingSize; ++i) {
        if (alpha[i] != 0) {
            coef.push_back(label[i] * alpha[i]);
            svIndex.push_back(i);
            
        }
    }
    checkCudaErrors(hipMemcpyFromSymbol(&rho, HIP_SYMBOL(devRho), sizeof(float_point), 0, hipMemcpyDeviceToHost));
    printf("# of SV %lu\nbias = %f\n", svIndex.size(), rho);
}

// void MultiSmoSolver::countPosiSV(const SvmProblem &subProblem, vector<int> &svIndex, vector<float_point> &coef,
//                                   float_point &rho, int posi_svcount) const {
// 	const unsigned int trainingSize = subProblem.getNumOfSamples();
// 	vector<float_point> alpha(trainingSize);
// 	const vector<int> &label = subProblem.v_nLabels;
// 	checkCudaErrors(hipMemcpy(alpha.data(), devAlpha, sizeof(float_point) * trainingSize, hipMemcpyDeviceToHost));
// 	for (int i = 0; i < trainingSize; ++i) {
// 		if (alpha[i] != 0) {
//             if(label[i]>0){ 
//                 posi_svcount++;
			
// 		}																        }
// 	checkCudaErrors(hipMemcpyFromSymbol(&rho, HIP_SYMBOL(devRho), sizeof(float_point), 0, hipMemcpyDeviceToHost));
// 	printf("# of SV %lu\nbias = %f\n", svIndex.size(), rho);
																	
// }
// }

MultiSmoSolver::MultiSmoSolver(const SvmProblem &problem, SvmModel &model, const SVMParam &param) :
        problem(problem), model(model), param(param) {
    q = 256;
    workingSetSize = 512;
    //workingSetSize must be 2^n and less than 1024
    assert(workingSetSize <= 1024);
    for (int i = 0; i < problem.getNumOfClasses(); ++i) {
        assert(workingSetSize <= problem.count[i]);
    }
    workingSet = vector<int>(workingSetSize);
    checkCudaErrors(hipMalloc((void **) &devAlphaDiff, sizeof(float_point) * workingSetSize));
    checkCudaErrors(hipMalloc((void **) &devWorkingSet, sizeof(int) * workingSetSize));
}

MultiSmoSolver::~MultiSmoSolver() {
    checkCudaErrors(hipFree(devAlphaDiff));
    checkCudaErrors(hipFree(devWorkingSet));
}

void MultiSmoSolver::selectWorkingSetAndPreCompute(const SvmProblem &subProblem, uint numOfSelectPairs) {
    uint numOfSamples = subProblem.getNumOfSamples();
    uint oldSize = workingSetSize / 2 - numOfSelectPairs;
    TIMER_START(selectTimer)
    thrust::device_ptr<float> valuePointer = thrust::device_pointer_cast(devFValue4Sort);
    thrust::device_ptr<int> indexPointer = thrust::device_pointer_cast(devIdx4Sort);
    vector<int> oldWorkingSet = workingSet;

    //get q most violation pairs
    getFUpValues << < gridSize, BLOCK_SIZE >> >
                                (devYiGValue, devAlpha, devLabel, numOfSamples, param.C, devFValue4Sort, devIdx4Sort);
    thrust::sort_by_key(valuePointer, valuePointer + numOfSamples, indexPointer, thrust::greater<float>());
    checkCudaErrors(hipMemcpy(workingSet.data() + oldSize * 2, devIdx4Sort, sizeof(int) * numOfSelectPairs,
                               hipMemcpyDeviceToHost));
    getFLowValues << < gridSize, BLOCK_SIZE >> >
                                 (devYiGValue, devAlpha, devLabel, numOfSamples, param.C, devFValue4Sort, devIdx4Sort);
    thrust::sort_by_key(valuePointer, valuePointer + numOfSamples, indexPointer, thrust::greater<float>());
    checkCudaErrors(
            hipMemcpy(workingSet.data() + oldSize * 2 + numOfSelectPairs, devIdx4Sort, sizeof(int) * numOfSelectPairs,
                       hipMemcpyDeviceToHost));

    //get pairs from last working set
    for (int i = 0; i < oldSize * 2; ++i) {
        workingSet[i] = oldWorkingSet[numOfSelectPairs * 2 + i];
    }
    checkCudaErrors(hipMemcpy(devWorkingSet, workingSet.data(), sizeof(int) * workingSetSize, hipMemcpyHostToDevice));
    TIMER_STOP(selectTimer)

    //move old kernel values to get space
    checkCudaErrors(hipMemcpy(devHessianMatrixCache,
                               devHessianMatrixCache + numOfSamples * numOfSelectPairs * 2,
                               sizeof(float_point) * numOfSamples * oldSize * 2,
                               hipMemcpyDeviceToDevice));
    vector<vector<KeyValue> > computeSamples;
    for (int i = 0; i < numOfSelectPairs * 2; ++i) {
        computeSamples.push_back(subProblem.v_vSamples[workingSet[oldSize * 2 + i]]);
    }
    TIMER_START(preComputeTimer)
    //preCompute kernel values of new selected instances
    hipsparseHandle_t handle;
    hipsparseMatDescr_t descr;
    CSRMatrix workingSetMat(computeSamples, subProblem.getNumOfFeatures());
    float_point * devWSVal;
    int *devWSColInd;
    int *devWSRowPtr;
    float_point * devWSSelfDot;
    workingSetMat.copy2Dev(devWSVal, devWSRowPtr, devWSColInd, devWSSelfDot);
    SubHessianCalculator::prepareCSRContext(handle, descr);
    CSRMatrix::CSRmm2Dense(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE, numOfSelectPairs * 2,
                           numOfSamples, subProblem.getNumOfFeatures(), descr,
                           workingSetMat.getNnz(), devWSVal, devWSRowPtr, devWSColInd, descr, nnz, devVal, devRowPtr,
                           devColInd, devHessianMatrixCache + numOfSamples * oldSize * 2);
    RBFKernel << < Ceil(numOfSelectPairs * 2 * numOfSamples, BLOCK_SIZE), BLOCK_SIZE >> > (devWSSelfDot, devSelfDot,
            devHessianMatrixCache + numOfSamples * oldSize * 2, numOfSelectPairs * 2, numOfSamples, param.gamma);
    SubHessianCalculator::releaseCSRContext(handle, descr);
    workingSetMat.freeDev(devWSVal, devWSRowPtr, devWSColInd, devWSSelfDot);
    TIMER_STOP(preComputeTimer)
}

