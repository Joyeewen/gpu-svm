#include "hip/hip_runtime.h"
/*
 * multiPredictor.cu
 *
 *  Created on: 1 Jan 2017
 *      Author: Zeyi Wen
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <assert.h>
#include "multiPredictor.h"
#include "predictionGPUHelper.h"
#include "classifierEvaluater.h"
#include "../svm-shared/constant.h"
#include "../SharedUtility/CudaMacro.h"
#include <iostream>
using namespace std;
real MultiPredictor::sigmoidPredict(real decValue, real A, real B) const {
    double fApB = decValue * A + B;
    // 1-p used later; avoid catastrophic cancellation
    if (fApB >= 0)
        return exp(-fApB) / (1.0 + exp(-fApB));
    else
        return 1.0 / (1 + exp(fApB));
}

void MultiPredictor::multiClassProbability(const vector<vector<real> > &r, vector<real> &p) const {
	int nrClass = model.nrClass;
    int t, j;
    int iter = 0, max_iter = max(100, nrClass);
    double **Q = (double **) malloc(sizeof(double *) * nrClass);
    double *Qp = (double *) malloc(sizeof(double) * nrClass);
    double pQp, eps = 0.005 / nrClass;

    for (t = 0; t < nrClass; t++) {
        p[t] = 1.0 / nrClass;  // Valid if k = 1
        Q[t] = (double *) malloc(sizeof(double) * nrClass);
        Q[t][t] = 0;
        for (j = 0; j < t; j++) {
            Q[t][t] += r[j][t] * r[j][t];
            Q[t][j] = Q[j][t];
        }
        for (j = t + 1; j < nrClass; j++) {
            Q[t][t] += r[j][t] * r[j][t];
            Q[t][j] = -r[j][t] * r[t][j];
        }
    }
    for (iter = 0; iter < max_iter; iter++) {
        // stopping condition, recalculate QP,pQP for numerical accuracy
        pQp = 0;
        for (t = 0; t < nrClass; t++) {
            Qp[t] = 0;
            for (j = 0; j < nrClass; j++)
                Qp[t] += Q[t][j] * p[j];
            pQp += p[t] * Qp[t];
        }
        double max_error = 0;
        for (t = 0; t < nrClass; t++) {
            double error = fabs(Qp[t] - pQp);
            if (error > max_error)
                max_error = error;
        }
        if (max_error < eps)
            break;

        for (t = 0; t < nrClass; t++) {
            double diff = (-Qp[t] + pQp) / Q[t][t];
            p[t] += diff;
            pQp = (pQp + diff * (diff * Q[t][t] + 2 * Qp[t])) / (1 + diff)
                  / (1 + diff);
            for (j = 0; j < nrClass; j++) {
                Qp[j] = (Qp[j] + diff * Q[t][j]) / (1 + diff);
                p[j] /= (1 + diff);
            }
        }
    }
    if (iter >= max_iter)
        printf("Exceeds max_iter in multiclass_prob\n");
    for (t = 0; t < nrClass; t++)
        free(Q[t]);
    free(Q);
    free(Qp);
}

vector<vector<real> > MultiPredictor::predictProbability(const vector<vector<KeyValue> > &v_vSamples, const vector<int> &vnOriginalLabel) const {
	int nrClass = model.nrClass;
    vector<vector<real> > result;
    vector<vector<real> > decValues;
    computeDecisionValues(v_vSamples, decValues);
    for (int l = 0; l < v_vSamples.size(); ++l) {
        vector<vector<real> > r(nrClass, vector<real>(nrClass));
        double min_prob = 1e-7;
        int k = 0;
        for (int i = 0; i < nrClass; i++)
            for (int j = i + 1; j < nrClass; j++) {
                r[i][j] = min(
                        max(sigmoidPredict(decValues[l][k], model.probA[k], model.probB[k]), min_prob), 1 - min_prob);
                r[j][i] = 1 - r[i][j];
                k++;
            }
        if(!vnOriginalLabel.empty())//want to measure sub-classifier error
        	ClassifierEvaluater::collectSubSVMInfo(model, l, vnOriginalLabel[l], nrClass, r, true);
        vector<real> p(nrClass);
        multiClassProbability(r, p);
        result.push_back(p);
    }
    return result;
}

/**
 * @brief: compute the decision value
 */
void MultiPredictor::computeDecisionValues(const vector<vector<KeyValue> > &v_vSamples,
                        		   vector<vector<real> > &decisionValues) const {
    //copy samples to device
    CSRMatrix sampleCSRMat(v_vSamples, model.numOfFeatures);
    real *devSampleVal;
    real *devSampleValSelfDot;
    int *devSampleRowPtr;
    int *devSampleColInd;
    int sampleNnz = sampleCSRMat.getNnz();
    checkCudaErrors(hipMalloc((void **) &devSampleVal, sizeof(real) * sampleNnz));
    checkCudaErrors(hipMalloc((void **) &devSampleValSelfDot, sizeof(real) * sampleCSRMat.getNumOfSamples()));
    checkCudaErrors(hipMalloc((void **) &devSampleRowPtr, sizeof(int) * (sampleCSRMat.getNumOfSamples() + 1)));
    checkCudaErrors(hipMalloc((void **) &devSampleColInd, sizeof(int) * sampleNnz));
    checkCudaErrors(hipMemcpy(devSampleVal, sampleCSRMat.getCSRVal(), sizeof(real) * sampleNnz,
                               hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(devSampleValSelfDot, sampleCSRMat.getCSRValSelfDot(),
                               sizeof(real) * sampleCSRMat.getNumOfSamples(), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(devSampleRowPtr, sampleCSRMat.getCSRRowPtr(),
    						   sizeof(int) * (sampleCSRMat.getNumOfSamples() + 1), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(devSampleColInd, sampleCSRMat.getCSRColInd(), sizeof(int) * sampleNnz,
    						   hipMemcpyHostToDevice));

    hipsparseHandle_t handle;
    hipsparseMatDescr_t descr;
    hipsparseCreate(&handle);
    hipsparseCreateMatDescr(&descr);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    real *devKernelValues;
    checkCudaErrors(hipMalloc((void **) &devKernelValues,
    						   sizeof(real) * v_vSamples.size() * model.svMap.size()));

    //dot product between sv and sample
    CSRMatrix::CSRmm2Dense(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
                           sampleCSRMat.getNumOfSamples(), model.svMapCSRMat->getNumOfSamples(),
						   model.svMapCSRMat->getNumOfFeatures(),
                           descr, sampleNnz, devSampleVal, devSampleRowPtr, devSampleColInd,
                           descr, model.svMapCSRMat->getNnz(), model.devSVMapVal, model.devSVMapRowPtr, model.devSVMapColInd,
                           devKernelValues);

    //obtain exp(-gamma*(a^2+b^2-2ab))
    int numOfBlock = Ceil(v_vSamples.size() * model.svMap.size(), BLOCK_SIZE);
    rbfKernel<<<numOfBlock, BLOCK_SIZE>>>(devSampleValSelfDot, sampleCSRMat.getNumOfSamples(),
                            		      model.devSVMapValSelfDot, model.svMapCSRMat->getNumOfSamples(),
										  devKernelValues, model.param.gamma);

    //sum kernel values of each model then obtain decision values
    int cnr2 = model.cnr2;
    numOfBlock = Ceil(v_vSamples.size() * cnr2, BLOCK_SIZE);
    real *devDecisionValues;
    checkCudaErrors(hipMalloc((void **) &devDecisionValues, sizeof(real) * v_vSamples.size() * cnr2));
    sumKernelValues<<<numOfBlock, BLOCK_SIZE>>>(devKernelValues, v_vSamples.size(),
    				model.svMapCSRMat->getNumOfSamples(), cnr2, model.devSVIndex,
					model.devCoef, model.devStart, model.devCount, model.devRho, devDecisionValues);
    real *tempDecValues = new real[v_vSamples.size() * cnr2];
    checkCudaErrors(hipMemcpy(tempDecValues, devDecisionValues,
                               sizeof(real) * v_vSamples.size() * cnr2, hipMemcpyDeviceToHost));
    decisionValues = vector<vector<real> >(v_vSamples.size(), vector<real>(cnr2));
    for (int i = 0; i < decisionValues.size(); ++i) {
        memcpy(decisionValues[i].data(), tempDecValues + i * cnr2, sizeof(real) * cnr2);
    }
    delete[] tempDecValues;
    checkCudaErrors(hipFree(devDecisionValues));
    checkCudaErrors(hipFree(devKernelValues));
    checkCudaErrors(hipFree(devSampleVal));
    checkCudaErrors(hipFree(devSampleValSelfDot));
    checkCudaErrors(hipFree(devSampleRowPtr));
    checkCudaErrors(hipFree(devSampleColInd));
    hipsparseDestroy(handle);
    hipsparseDestroyMatDescr(descr);
}

/**
 * @brief: predict the label of the instances
 * @param: vnOriginalLabel is for computing errors of sub-classifier.
 */
vector<int> MultiPredictor::predict(const vector<vector<KeyValue> > &v_vSamples, const vector<int> &vnOriginalLabel) const{
	int nrClass = model.nrClass;
    int manyClassIns=0;
	bool probability = model.isProbability();
    vector<int> labels;
    if (!probability) {
        vector<vector<real> > decisionValues;
        computeDecisionValues(v_vSamples, decisionValues);
			cout<<"sample "<<v_vSamples[0][0].featureValue<<endl;
			cout<<"sample "<<v_vSamples[0][1].featureValue<<endl;
			cout<<"sample "<<v_vSamples[0][2].featureValue<<endl;
        for (int l = 0; l < v_vSamples.size(); ++l) {
        	if(!vnOriginalLabel.empty())//want to measure sub-classifier error
	            ClassifierEvaluater::collectSubSVMInfo(model, l, vnOriginalLabel[l], nrClass, decisionValues, false);

            vector<int> votes(nrClass, 0);
            int k = 0;
            for (int i = 0; i < nrClass; ++i) {
                for (int j = i + 1; j < nrClass; ++j) {
		    if(l<1){
		       cout<<"gpu decisionvalue for 1 instance "<<decisionValues[l][k]<<endl;
 		    }
                    if (decisionValues[l][k++] > 0)
                    	votes[i]++;
                    else
                    	votes[j]++;
                }
            }
            int maxVoteClass = 0;
            for (int i = 0; i < nrClass; ++i) {
                if (votes[i] > votes[maxVoteClass])
                    maxVoteClass = i;
            }
            labels.push_back(model.label[maxVoteClass]);
	    if(l<20){
	        cout<<"****************predict 10 label"<<endl;
		//cout<<"maxvote "<<maxVoteClass<<endl;
		//cout<<"max label "<<model.label[maxVoteClass]<<endl;
		}
            //compute #instance that belong to more than one classes
            int flag=0;
            for(int i=0;i<nrClass;i++)
                for(int j=i+1;j<nrClass;j++){
                    if(votes[i]==votes[j]){
                        flag++;
                        break;
                    }
                }
            if(flag>0)
                manyClassIns++;
        }
       // printf("number of instance belong to manyClass %.2f%%%(%d,%d)\n",manyClassIns/ (float) v_vSamples.size(), manyClassIns,v_vSamples.size());
    } else {
        assert(model.probability);
        vector<vector<real> > prob = predictProbability(v_vSamples, vnOriginalLabel);
        // todo select max using GPU
        for (int i = 0; i < v_vSamples.size(); ++i) {
            int maxProbClass = 0;
            for (int j = 0; j < nrClass; ++j) {
                if (prob[i][j] > prob[i][maxProbClass])
                    maxProbClass = j;
            }
            labels.push_back(model.label[maxProbClass]);
        }
    }
    return labels;
}

void MultiPredictor::predictDecValue(vector<real> &combDecValue, const vector<vector<KeyValue> > &v_vSamples) const{
    int nrClass = model.nrClass;

        vector<vector<real> > decisionValues;
        computeDecisionValues(v_vSamples, decisionValues);

        for (int l = 0; l < v_vSamples.size(); ++l) {
            combDecValue.push_back(decisionValues[l][0]);

        }

}
